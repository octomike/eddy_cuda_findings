//Example 1. Application Using C and cuBLAS: 1-based indexing
//-----------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define K 3 //1196032
#define N 3 //54
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float *devPtrA, *devPtrC;
    float *a=0, *c=0;
    float alpha = 1.0, beta = 1.0;

    a = (float *)malloc (N * K * sizeof (*a));
    c = (float *)malloc (N * N * sizeof (*c));
    if (!a || !c) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    for (j = 1; j <= N; j++) {
        for (i = 1; i <= K; i++) {
            a[IDX2F(i,j,K)] = 1.0f;
        }
    }
    for (j = 1; j <= N; j++) {
        for (i = 1; i <= N; i++) {
            c[IDX2F(i,j,N)] = 1.0f;
        }
    }
    cudaStat = hipMalloc ((void**)&devPtrA, N*K*sizeof(*a));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed for A");
        return EXIT_FAILURE;
    }
    cudaStat = hipMalloc ((void**)&devPtrC, N*N*sizeof(*c));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed for C");
        return EXIT_FAILURE;
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (K, N, sizeof(*a), a, K, devPtrA, K);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (devPtrA);
        hipFree (devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (N, N, sizeof(*c), c, N, devPtrC, N);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (devPtrA);
        hipFree (devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    stat = hipblasSsyrk (handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, N, K, &alpha, devPtrA, K, &beta, devPtrC, N);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("%s: %s\n", "hipblasSsyrk failed", _cudaGetErrorEnum(stat));
        hipFree (devPtrA);
        hipFree (devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    stat = hipblasGetMatrix (N, N, sizeof(*c), devPtrC, N, c, N);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipFree (devPtrC);
    hipblasDestroy(handle);
    for (j = 1; j <= N; j++) {
        for (i = 1; i <= j; i++) {
            printf ("%7.0f", c[IDX2F(i,j,N)]);
        }
        printf ("\n");
    }
    free(a);
    free(c);
    return EXIT_SUCCESS;
}
